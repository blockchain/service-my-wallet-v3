#include "hip/hip_runtime.h"
//
//  =============== BLAKE part on nVidia GPU ======================
//
// This is the generic "default" implementation when no architecture
// specific implementation is available in the kernel.
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//
// TODO: CUDA porting work remains to be done.
//

#include <map>
#include <stdint.h>

#include "salsa_kernel.h"
#include "miner.h"

typedef uint32_t sph_u32;
#define SPH_C32(x) ((sph_u32)(x))
#define SPH_T32(x) ((x) & SPH_C32(0xFFFFFFFF))
#define SPH_ROTL32(x, n)   SPH_T32(((x) << (n)) | ((x) >> (32 - (n))))
#define SPH_ROTR32(x, n)   SPH_ROTL32(x, (32 - (n)))

__constant__ uint64_t ptarget64[4];
__constant__ uint32_t pdata[20];

// define some error checking macros
#undef checkCudaErrors

#if WIN32
#define DELIMITER '/'
#else
#define DELIMITER '/'
#endif
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) calling '%s' (%s line %d)\n", device_map[thr_id], err, hipGetErrorString(err), #x, __FILENAME__, __LINE__); \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

#ifdef _MSC_VER
#pragma warning (disable: 4146)
#endif

static __device__ sph_u32 cuda_sph_bswap32(sph_u32 x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

/**
 * Encode a 32-bit value into the provided buffer (big endian convention).
 *
 * @param dst   the destination buffer
 * @param val   the 32-bit value to encode
 */
static __device__ void
cuda_sph_enc32be(void *dst, sph_u32 val)
{
    *(sph_u32 *)dst = cuda_sph_bswap32(val);
}

#define Z00   0
#define Z01   1
#define Z02   2
#define Z03   3
#define Z04   4
#define Z05   5
#define Z06   6
#define Z07   7
#define Z08   8
#define Z09   9
#define Z0A   A
#define Z0B   B
#define Z0C   C
#define Z0D   D
#define Z0E   E
#define Z0F   F

#define Z10   E
#define Z11   A
#define Z12   4
#define Z13   8
#define Z14   9
#define Z15   F
#define Z16   D
#define Z17   6
#define Z18   1
#define Z19   C
#define Z1A   0
#define Z1B   2
#define Z1C   B
#define Z1D   7
#define Z1E   5
#define Z1F   3

#define Z20   B
#define Z21   8
#define Z22   C
#define Z23   0
#define Z24   5
#define Z25   2
#define Z26   F
#define Z27   D
#define Z28   A
#define Z29   E
#define Z2A   3
#define Z2B   6
#define Z2C   7
#define Z2D   1
#define Z2E   9
#define Z2F   4

#define Z30   7
#define Z31   9
#define Z32   3
#define Z33   1
#define Z34   D
#define Z35   C
#define Z36   B
#define Z37   E
#define Z38   2
#define Z39   6
#define Z3A   5
#define Z3B   A
#define Z3C   4
#define Z3D   0
#define Z3E   F
#define Z3F   8

#define Z40   9
#define Z41   0
#define Z42   5
#define Z43   7
#define Z44   2
#define Z45   4
#define Z46   A
#define Z47   F
#define Z48   E
#define Z49   1
#define Z4A   B
#define Z4B   C
#define Z4C   6
#define Z4D   8
#define Z4E   3
#define Z4F   D

#define Z50   2
#define Z51   C
#define Z52   6
#define Z53   A
#define Z54   0
#define Z55   B
#define Z56   8
#define Z57   3
#define Z58   4
#define Z59   D
#define Z5A   7
#define Z5B   5
#define Z5C   F
#define Z5D   E
#define Z5E   1
#define Z5F   9

#define Z60   C
#define Z61   5
#define Z62   1
#define Z63   F
#define Z64   E
#define Z65   D
#define Z66   4
#define Z67   A
#define Z68   0
#define Z69   7
#define Z6A   6
#define Z6B   3
#define Z6C   9
#define Z6D   2
#define Z6E   8
#define Z6F   B

#define Z70   D
#define Z71   B
#define Z72   7
#define Z73   E
#define Z74   C
#define Z75   1
#define Z76   3
#define Z77   9
#define Z78   5
#define Z79   0
#define Z7A   F
#define Z7B   4
#define Z7C   8
#define Z7D   6
#define Z7E   2
#define Z7F   A

#define Z80   6
#define Z81   F
#define Z82   E
#define Z83   9
#define Z84   B
#define Z85   3
#define Z86   0
#define Z87   8
#define Z88   C
#define Z89   2
#define Z8A   D
#define Z8B   7
#define Z8C   1
#define Z8D   4
#define Z8E   A
#define Z8F   5

#define Z90   A
#define Z91   2
#define Z92   8
#define Z93   4
#define Z94   7
#define Z95   6
#define Z96   1
#define Z97   5
#define Z98   F
#define Z99   B
#define Z9A   9
#define Z9B   E
#define Z9C   3
#define Z9D   C
#define Z9E   D
#define Z9F   0

#define Mx(r, i)    Mx_(Z ## r ## i)
#define Mx_(n)      Mx__(n)
#define Mx__(n)     M ## n

#define CSx(r, i)   CSx_(Z ## r ## i)
#define CSx_(n)     CSx__(n)
#define CSx__(n)    CS ## n

#define CS0   SPH_C32(0x243F6A88)
#define CS1   SPH_C32(0x85A308D3)
#define CS2   SPH_C32(0x13198A2E)
#define CS3   SPH_C32(0x03707344)
#define CS4   SPH_C32(0xA4093822)
#define CS5   SPH_C32(0x299F31D0)
#define CS6   SPH_C32(0x082EFA98)
#define CS7   SPH_C32(0xEC4E6C89)
#define CS8   SPH_C32(0x452821E6)
#define CS9   SPH_C32(0x38D01377)
#define CSA   SPH_C32(0xBE5466CF)
#define CSB   SPH_C32(0x34E90C6C)
#define CSC   SPH_C32(0xC0AC29B7)
#define CSD   SPH_C32(0xC97C50DD)
#define CSE   SPH_C32(0x3F84D5B5)
#define CSF   SPH_C32(0xB5470917)

#define GS(m0, m1, c0, c1, a, b, c, d)   do { \
        a = SPH_T32(a + b + (m0 ^ c1)); \
        d = SPH_ROTR32(d ^ a, 16); \
        c = SPH_T32(c + d); \
        b = SPH_ROTR32(b ^ c, 12); \
        a = SPH_T32(a + b + (m1 ^ c0)); \
        d = SPH_ROTR32(d ^ a, 8); \
        c = SPH_T32(c + d); \
        b = SPH_ROTR32(b ^ c, 7); \
    } while (0)

#define ROUND_S(r)   do { \
        GS(Mx(r, 0), Mx(r, 1), CSx(r, 0), CSx(r, 1), V0, V4, V8, VC); \
        GS(Mx(r, 2), Mx(r, 3), CSx(r, 2), CSx(r, 3), V1, V5, V9, VD); \
        GS(Mx(r, 4), Mx(r, 5), CSx(r, 4), CSx(r, 5), V2, V6, VA, VE); \
        GS(Mx(r, 6), Mx(r, 7), CSx(r, 6), CSx(r, 7), V3, V7, VB, VF); \
        GS(Mx(r, 8), Mx(r, 9), CSx(r, 8), CSx(r, 9), V0, V5, VA, VF); \
        GS(Mx(r, A), Mx(r, B), CSx(r, A), CSx(r, B), V1, V6, VB, VC); \
        GS(Mx(r, C), Mx(r, D), CSx(r, C), CSx(r, D), V2, V7, V8, VD); \
        GS(Mx(r, E), Mx(r, F), CSx(r, E), CSx(r, F), V3, V4, V9, VE); \
    } while (0)

#define COMPRESS32   do { \
        sph_u32 M0, M1, M2, M3, M4, M5, M6, M7; \
        sph_u32 M8, M9, MA, MB, MC, MD, ME, MF; \
        sph_u32 V0, V1, V2, V3, V4, V5, V6, V7; \
        sph_u32 V8, V9, VA, VB, VC, VD, VE, VF; \
        V0 = H0; \
        V1 = H1; \
        V2 = H2; \
        V3 = H3; \
        V4 = H4; \
        V5 = H5; \
        V6 = H6; \
        V7 = H7; \
        V8 = S0 ^ CS0; \
        V9 = S1 ^ CS1; \
        VA = S2 ^ CS2; \
        VB = S3 ^ CS3; \
        VC = T0 ^ CS4; \
        VD = T0 ^ CS5; \
        VE = T1 ^ CS6; \
        VF = T1 ^ CS7; \
        M0 = input[0]; \
        M1 = input[1]; \
        M2 = input[2]; \
        M3 = input[3]; \
        M4 = input[4]; \
        M5 = input[5]; \
        M6 = input[6]; \
        M7 = input[7]; \
        M8 = input[8]; \
        M9 = input[9]; \
        MA = input[10]; \
        MB = input[11]; \
        MC = input[12]; \
        MD = input[13]; \
        ME = input[14]; \
        MF = input[15]; \
        ROUND_S(0); \
        ROUND_S(1); \
        ROUND_S(2); \
        ROUND_S(3); \
        ROUND_S(4); \
        ROUND_S(5); \
        ROUND_S(6); \
        ROUND_S(7); \
        H0 ^= S0 ^ V0 ^ V8; \
        H1 ^= S1 ^ V1 ^ V9; \
        H2 ^= S2 ^ V2 ^ VA; \
        H3 ^= S3 ^ V3 ^ VB; \
        H4 ^= S0 ^ V4 ^ VC; \
        H5 ^= S1 ^ V5 ^ VD; \
        H6 ^= S2 ^ V6 ^ VE; \
        H7 ^= S3 ^ V7 ^ VF; \
    } while (0)

__global__ void cuda_blake256_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate )
{
    uint32_t input[16];
    uint64_t output[4];

#pragma unroll 16
    for (int i=0; i < 16; ++i) input[i] = pdata[i];

    sph_u32 H0 = 0x6A09E667;
    sph_u32 H1 = 0xBB67AE85;
    sph_u32 H2 = 0x3C6EF372;
    sph_u32 H3 = 0xA54FF53A;
    sph_u32 H4 = 0x510E527F;
    sph_u32 H5 = 0x9B05688C;
    sph_u32 H6 = 0x1F83D9AB;
    sph_u32 H7 = 0x5BE0CD19;
    sph_u32 S0 = 0;
    sph_u32 S1 = 0;
    sph_u32 S2 = 0;
    sph_u32 S3 = 0;
    sph_u32 T0 = 0;
    sph_u32 T1 = 0;
    T0 = SPH_T32(T0 + 512);
    COMPRESS32;

#pragma unroll 3
    for (int i=0; i < 3; ++i) input[i] = pdata[16+i];
    input[3] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
    input[4] = 0x80000000;
#pragma unroll 8
    for (int i=5; i < 13; ++i) input[i] = 0;
    input[13] = 0x00000001;
    input[14] = T1;
    input[15] = T0 + 128;

    T0 = SPH_T32(T0 + 128);
    COMPRESS32;

    cuda_sph_enc32be((unsigned char*)output + 4*6, H6);
    cuda_sph_enc32be((unsigned char*)output + 4*7, H7);
    if (validate || output[3] <=  ptarget64[3])
    {
        // this data is only needed when we actually need to save the hashes
        cuda_sph_enc32be((unsigned char*)output + 4*0, H0);
        cuda_sph_enc32be((unsigned char*)output + 4*1, H1);
        cuda_sph_enc32be((unsigned char*)output + 4*2, H2);
        cuda_sph_enc32be((unsigned char*)output + 4*3, H3);
        cuda_sph_enc32be((unsigned char*)output + 4*4, H4);
        cuda_sph_enc32be((unsigned char*)output + 4*5, H5);
    }

    if (validate)
    {
        g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
#pragma unroll 4
        for (int i=0; i < 4; ++i) g_out[i] = output[i];
    }

    if (output[3] <=  ptarget64[3]) {
        uint64_t *g_good64 = (uint64_t*)g_good;
        if (output[3] < g_good64[3]) {
            g_good64[3] = output[3];
            g_good64[2] = output[2];
            g_good64[1] = output[1];
            g_good64[0] = output[0];
            g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
        }
    }
}

static std::map<int, uint32_t *> context_good[2];

extern "C" void default_prepare_blake256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        // allocate pinned host memory for good hashes
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

        init[thr_id] = true;
    }
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

extern "C" bool default_do_blake256(dim3 grid, dim3 threads, int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
    bool success = true;
  
    checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

    cuda_blake256_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

    // copy hashes from device memory to host (ALL hashes, lots of data...)
    if (do_d2h && hash != NULL) {
        size_t mem_size = throughput * sizeof(uint32_t) * 8;
        checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }
    else if (hash != NULL) {
        // asynchronous copy of winning nonce (just 4 bytes...)
        checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }

        // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}
